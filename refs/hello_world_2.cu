
#include <hip/hip_runtime.h>
#include <stdio.h>
#define SIZE 1024

__global__ void VectorAdd(int *a, int *b, int *c, int n)
{
    int i = threadIdx.x;

    if (i < n)
        c[i] = a[i] + b[i];
}

int *a, *b, *c;

int main()
{
    // init
    a = (int *)malloc(SIZE * sizeof(int));
    b = (int *)malloc(SIZE * sizeof(int));
    c = (int *)malloc(SIZE * sizeof(int));
    for (int i = 0; i < SIZE; ++i)
    {
        a[i] = i;
        b[i] = i;
        c[i] = 0;
    }

    // cuda init
    int *ad, *bd, *cd;    
    int size = SIZE * sizeof(int);
    hipMalloc(&ad, size);
    hipMemcpy(ad, a, size, hipMemcpyHostToDevice);
    hipMalloc(&bd, size);
    hipMemcpy(bd, b, size, hipMemcpyHostToDevice);
    hipMalloc(&cd, size);

    // cuda execute
    int dimGrid = 1;
    int dimBlock = SIZE;
    VectorAdd <<<dimGrid, dimBlock>>> (ad, bd, cd, SIZE);
    hipDeviceSynchronize();

    // cuda result
    hipMemcpy(c, cd, size, hipMemcpyDeviceToHost);
    for (int i = 0; i < 10; ++i)
        printf("c[%d] = %d\n", i, c[i]);

    // cuda free
    hipFree(ad);
    hipFree(bd);
    hipFree(cd);

    return 0;
}
